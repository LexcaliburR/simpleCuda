#include "hip/hip_runtime.h"
#include <chrono>
#include <thread>
#include <hip/hip_runtime.h>
#include <iostream>
#include "common/check_utils.h"

__global__ void ArrayAddV2(float* array1, float* array2, float* result) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    result[idx] = array1[idx] + array2[idx];
    printf("%f", result[idx]);
    return;
}

int main()
{
    std::cout << "1111111111111" << std::endl;
    size_t size = 10000;
    int loop_times = 100;
    float* array1_h = new float[size];
    float* array2_h = new float[size];
    float* result_h = new float[size];
    for(int i = 0; i < size; ++i) {
        array1_h[i] = 1.1f;
        array2_h[i] = 2.2f;
        result_h[i] = 0.f;
    }

    float* array1_d = nullptr;
    float* array2_d = nullptr;
    float* result_d = nullptr;
    //    CUDA_CHECK(hipMalloc((void**)&array1_d, size * sizeof(float)));
    //    CUDA_CHECK();
    hipMalloc((void**)&array1_d, size * sizeof(float));
//    CUDA_CHECK(hipMalloc((void**)&array2_d, size * sizeof(float)));
//    CUDA_CHECK(hipMalloc((void**)&result_d, size * sizeof(float)));
//    CUDA_CHECK(hipMemcpy(array1_d, array1_h, size * sizeof(float), hipMemcpyHostToDevice));
//    CUDA_CHECK(hipMemcpy(array2_d, array2_h, size * sizeof(float), hipMemcpyHostToDevice));
//    CUDA_CHECK(hipMemset(result_d, 0, size * sizeof(float)));
    hipMalloc((void**)&array2_d, size * sizeof(float));
    hipMalloc((void**)&result_d, size * sizeof(float));
    hipMemcpy(array1_d, array1_h, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(array2_d, array2_h, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(result_d, 0, size * sizeof(float));

    for(int j = 0; j < loop_times; j++) {
        ArrayAddV2<<<(size + 255) / 256, 256>>>(array1_d, array2_d, result_d);
    }
    hipDeviceSynchronize();
//    delete[] array1_h;
//    delete[] array2_h;
//    delete[] result_h;
//    CUDA_CHECK(hipFree(array1_d));
//    CUDA_CHECK(hipFree(array2_d));
//    CUDA_CHECK(hipFree(result_d));
    return 0;
}